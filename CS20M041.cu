#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>

using namespace std;

__global__ void fill(int *p, int *train, int size){

	int id = blockIdx.x * 1024 + threadIdx.x;

	if(id<size){
		for(int i=0; i<25; i++){
			for(int j=0; j<50; j++){
				p[id*25*50 + i*50 + j] = train[id*28+3+i];
			}
		}
	}
}

__global__ void fill1(int reqst, int *p, int *q){

	int id = blockIdx.x*1024+threadIdx.x;
	if(id<reqst){
		p[id]=0;
		q[id]=0;
	}
}

__global__ void book_tickets(int Request, int *seats, int *request, int *sorted, int *train_No, int *coach, int *src, int *dest, int *tickets, int *train_coach, int *train_info, int *result, int *booked, int *free)
{
	int id = blockIdx.x * 1024 + threadIdx.x;
	if(id<Request)
	{

		int reqst, Source, Destination, temp, m, dist, train, train_src;
		//booked[id]=0;
		//result[id]=0;
		reqst = sorted[id];
		if(src[reqst]>dest[reqst]){
			temp = src[reqst];
			src[reqst] = dest[reqst];
			dest[reqst] = temp;
		}
		Source = src[reqst];
		m = coach[reqst];
		train = train_No[reqst];
		Destination = dest[reqst];
		train_src = train_info[train*28];

		if(id==0)
		{
			for(int i=Source; i<Destination; i++){
				if(free[train*25*50 + m*50 +(i-train_src)]<tickets[reqst]){
					booked[id] = 1;
					break;
				}
			}

			if(!booked[id]){
				for(int i = Source; i<Destination; i++){
					free[train*25*50 + m*50 + (i-train_src)]-=tickets[reqst];
				}
				result[reqst] = 1;
				booked[id] = 1;
			}
			booked[id]=1;
		}
		else
		{
			do
			{
				if((train_coach[id]!=train_coach[id-1])||(booked[id-1]))
				{
					for(int i=Source; i<Destination; i++){
						if(free[train*25*50 + m*50 +(i-train_src)]<tickets[reqst]){
							booked[id] = 1;
							break;
						}
					}

					if(!booked[id]){
						for(int i = Source; i<Destination; i++){
							free[train*25*50 + m*50 + (i-train_src)]-=tickets[reqst];
						}
						result[reqst] = 1;
						booked[id] = 1;
					}

					booked[id] = 1;
				}
			}while(!booked[id]);
		}

		if(result[reqst]){
			dist = tickets[reqst]*(Destination - Source);
			atomicAdd(seats,dist);
		}
	}
}

void writeResult(int *result, int request){

	int success_count=0, failure_count = 0;
	for (int i = 0; i < request; ++i)
	{
		if(result[i]){
			cout<<"success"<<"\n";
			success_count+=1;
		}
		else{
			cout<<"failure"<<"\n";
			failure_count+=1;
		}
	}
	cout<<success_count<<" "<<failure_count<<"\n";
}

int main(){

	int N, M, src, dest, train, temp;
	int *train_info, *Dtrain_info, *DTemp;
	int *Result, *DBooked, *DResult;
	int batch, request, *seats;
	int *Request, *Train, *Coach, *Source, *Dest, *Tickets, *Train_coach;
	int *DRequest, *DTrain, *DCoach, *DSource, *DDest, *DTickets, *DTrain_coach;
	
	cin>>N;
	train_info = (int *)malloc(N*28*sizeof(int));
	for (int i = 0; i < N; ++i)
	{
		cin>>train>>M>>src>>dest;
		if(src>dest){
			temp = src;
			src = dest;
			dest = temp;
		}
		train_info[i*28] = src;
		train_info[i*28+1] = dest;
		train_info[i*28+2] = M;
		for(int j=0; j<M; j++){
			cin>>temp>>train_info[i*28+3+j];
		}
		for(int k=M; k<25; k++){
			train_info[i*28+3+k] = 0;
		}
	}

	hipMalloc(&Dtrain_info, N*28*sizeof(int));
	hipMemcpy(Dtrain_info, train_info, N*28*sizeof(int), hipMemcpyHostToDevice);
	int *Free = (int *)malloc(N*25*50*sizeof(int)), *Dfree;
	hipMalloc(&Dfree, N*25*50*sizeof(int));
	fill<<<(N+1023)/1024, 1024>>>(Dfree, Dtrain_info, N);
	
	cin>>batch;
	for (int i = 0; i < batch; ++i)
	{
		cin>>request;
		Request = (int *)malloc(request*sizeof(int));
		Train = (int *)malloc(request*sizeof(int));
		Coach = (int *)malloc(request*sizeof(int));
		Source = (int *)malloc(request*sizeof(int));
		Dest = (int *)malloc(request*sizeof(int));
		Tickets = (int *)malloc(request*sizeof(int));
		Train_coach = (int *)malloc(request*sizeof(int));
		for(int j=0; j<request; j++){

			cin>>Request[j]>>Train[j]>>Coach[j]>>Source[j]>>Dest[j]>>Tickets[j];
			string str = "0", str1 = to_string(Train[j]), str2 = to_string(Coach[j]),s;

			if((Coach[j]/10)==0){
				s=str+str2;
			}
			else{
				s=str2;
			}
			Train_coach[j] = stoi(str1+s);
		}

		hipMalloc(&DRequest, request*sizeof(int));
		hipMalloc(&DTrain, request*sizeof(int));
		hipMalloc(&DCoach, request*sizeof(int));
		hipMalloc(&DSource, request*sizeof(int));
		hipMalloc(&DDest, request*sizeof(int));
		hipMalloc(&DTickets, request*sizeof(int));
		hipMalloc(&DTrain_coach, request*sizeof(int));
		hipMalloc(&DTemp, request*sizeof(int));
		hipMalloc(&DBooked, request*sizeof(int));
		hipMalloc(&DResult, request*sizeof(int));
		hipHostAlloc(&seats, sizeof(int), 0);

		hipMemcpy(DRequest, Request, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DTemp, Request, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DTrain, Train, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DCoach, Coach, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DSource, Source, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DDest, Dest, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DTickets, Tickets, request*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(DTrain_coach, Train_coach, request*sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		
		thrust::sort_by_key(thrust::device, DTrain_coach, DTrain_coach + request, DTemp);
		hipDeviceSynchronize();
		fill1<<<(request+1023)/1024, 1024>>>(request, DBooked, DResult);
		book_tickets<<<(request+1023)/1024, 1024>>>(request, seats, DRequest, DTemp, DTrain, DCoach, DSource, DDest, DTickets, DTrain_coach, Dtrain_info, DResult, DBooked, Dfree);
		free(Request);
		free(Train);
		free(Coach);
		free(Source);
		free(Dest);
		free(Tickets);
		free(Train_coach);
		Result = (int *)malloc(request * sizeof(int));
		hipDeviceSynchronize();
		hipMemcpy(Result, DResult, request*sizeof(int), hipMemcpyDeviceToHost);
		
		hipFree(DRequest);
		hipFree(DTrain);
		hipFree(DCoach);
		hipFree(DSource);
		hipFree(DDest);
		hipFree(DTickets);
		hipFree(DTrain_coach);
		hipFree(DTemp);
		hipFree(DBooked);
		hipFree(DResult);
		writeResult(Result, request);
		cout<<*seats<<"\n";
		free(Result);
	}

	hipFree(Dtrain_info);
	hipFree(Dfree);
	free(train_info);
	free(Free);
	return 0;
}